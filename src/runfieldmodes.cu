#include "hip/hip_runtime.h"
/*
bellhopcxx / bellhopcuda - C++/CUDA port of BELLHOP underwater acoustics simulator
Copyright (C) 2021-2022 The Regents of the University of California
c/o Jules Jaffe team at SIO / UCSD, jjaffe@ucsd.edu
Based on BELLHOP, which is Copyright (C) 1983-2020 Michael B. Porter

This program is free software: you can redistribute it and/or modify it under
the terms of the GNU General Public License as published by the Free Software
Foundation, either version 3 of the License, or (at your option) any later
version.

This program is distributed in the hope that it will be useful, but WITHOUT ANY
WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
PARTICULAR PURPOSE. See the GNU General Public License for more details.

You should have received a copy of the GNU General Public License along with
this program. If not, see <https://www.gnu.org/licenses/>.
*/
#include "run.hpp"

namespace bhc {

#define NUM_THREADS 256
#define LAUNCH_BOUNDS __launch_bounds__(NUM_THREADS, 1)

template<bool O3D, bool R3D> __global__ void LAUNCH_BOUNDS
FieldModesKernel(bhcParams<O3D, R3D> params, bhcOutputs<O3D, R3D> outputs)
{
    for(int32_t job = blockIdx.x * blockDim.x + threadIdx.x; true;
        job += gridDim.x * blockDim.x) {
        RayInitInfo rinit;
        if(!GetJobIndices<O3D>(rinit, job, params.Pos, params.Angles)) break;

        MainFieldModes<O3D, R3D>(
            rinit, outputs.uAllSources, params.Bdry, params.bdinfo, params.refl,
            params.ssp, params.Pos, params.Angles, params.freqinfo, params.Beam,
            params.beaminfo, outputs.eigen, outputs.arrinfo);
    }
}

template<bool O3D, bool R3D> void RunFieldModesImpl(
    bhcParams<O3D, R3D> &params, bhcOutputs<O3D, R3D> &outputs, uint32_t cores)
{
    IGNORE_UNUSED(cores);
    FieldModesKernel<O3D, R3D><<<d_multiprocs, NUM_THREADS>>>(params, outputs);
    syncAndCheckKernelErrors("FieldModesKernel");
}

#if BHC_ENABLE_2D
template void RunFieldModesImpl<false, false>(
    bhcParams<false, false> &params, bhcOutputs<false, false> &outputs, uint32_t cores);
#endif
#if BHC_ENABLE_NX2D
template void RunFieldModesImpl<true, false>(
    bhcParams<true, false> &params, bhcOutputs<true, false> &outputs, uint32_t cores);
#endif
#if BHC_ENABLE_3D
template void RunFieldModesImpl<true, true>(
    bhcParams<true, true> &params, bhcOutputs<true, true> &outputs, uint32_t cores);
#endif

} // namespace bhc
